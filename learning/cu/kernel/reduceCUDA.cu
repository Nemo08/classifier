// reduce_kernel.cu
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>

// Device hash function (adapted from the Go version)
__device__ uint32_t hash_d(uint32_t n, uint32_t salt, uint32_t maxx) {
    uint32_t m = n - salt;
    m ^= m << 2;
    m ^= m << 3;
    m ^= m >> 5;
    m ^= m >> 7;
    m ^= m << 11;
    m ^= m << 13;
    m ^= m >> 17;
    m ^= m << 19;
    m += salt;
    return (uint32_t)(((uint64_t)m * maxx) >> 32);
}

/*
  d_nums is expected to have the following layout:
    d_nums[0] : maxx
    d_nums[1] : len0  (current length of alphabet0)
    d_nums[2] : len1  (current length of alphabet1)
    d_nums[3] : DeadlineMs
    d_nums[4] : tasks (how many parallel entities should run)
    d_nums[5] : iteration (current iteration; may be used for exit checking)
    d_nums[6] : center
    d_nums[7] : out_offset  (used to assign program entries; each entry takes 2 uint32_t)
    d_nums[8] : minsub
    d_nums[9] : res (number of result tuples which can be stored in out)
    d_nums[10]: minadd
    d_nums[11]: arenasize
    d_nums[12]: mutex
    d_nums[13]: mustExit
    d_nums[14]: allocator
    d_nums[15]: subtractor
*/
// Declare the mutex variable in device memory.
// Mark it as volatile to prevent unwanted compiler optimizations.

// Declare the mutex variable in device memory.
// Mark it as volatile to prevent unwanted compiler optimizations.

__device__ static uint32_t hashCounter = 0;

__device__ static uint32_t* alpha0 = 0;
__device__ static uint32_t* alpha1 = 0;
///__device__ static uint32_t* free0 = 0;
///__device__ static uint32_t* free1 = 0;

// Full kernel definition with no omitted parts.
extern "C" __global__ void reduce(uint8_t *d_set, uint32_t *d_nums,
                                    uint32_t *alphabet0, uint32_t *alphabet1,
                                    uint32_t* arena, uint32_t* out) {

	// Lock the mutex: attempt to acquire the lock.
	auto lockMutex = [&] __device__ () -> int {
		
	    // Try to acquire the lock by comparing the value at the address of globalMutex with 0.
	    // If it is 0, then set it to 1 and the lock is acquired.
	    if (atomicCAS((uint32_t *)(void *)&d_nums[12], 0, 1) != 0) {
			if (atomicCAS((uint32_t *)(void *)&d_nums[12], 0, 1) != 0) {
				if (atomicCAS((uint32_t *)(void *)&d_nums[12], 0, 1) != 0) {
					return 1;
				}
			}
		}
	    // Memory fence to ensure subsequent memory accesses see updated values.
	    
	    return 0;
	};

	// Unlock the mutex.
	auto unlockMutex = [&] __device__ () -> void {
	    // Memory fence to ensure all previous writes are visible.
	    
	    atomicExch((uint32_t *)&d_nums[12], 0);
	    
	};

        uint32_t iteration = d_nums[5];  // not used explicitly for exit here
    // Compute a unique thread ID (we assume a full grid launch)
	int myFlag = iteration;

	// Must exit.
	auto mustExit = [&] __device__ () -> int {
	    // Memory fence to ensure all previous writes are visible.
	    
	    int ret = atomicAdd((uint32_t *)(void *)&d_nums[13], 0) > myFlag;
	    
	    return ret;
	};


	alpha0 = alphabet0;
	alpha1 = alphabet1;

	auto allocate = [&] __device__ (uint32_t sizee) -> uint32_t {
		return atomicAdd((uint32_t *)(void *)&d_nums[14], sizee);
	};




        //uint32_t maxmax      = d_nums[0];
        uint32_t scratch      = (((d_nums[0] + 3) / 4) + 4);

	uint64_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
	uint64_t tid_z = blockIdx.z * blockDim.z + threadIdx.z;

	uint64_t numThreads_x = gridDim.x * blockDim.x;
	uint64_t numThreads_y = gridDim.y * blockDim.y;

	uint64_t tid = tid_x + tid_y * numThreads_x + tid_z * (numThreads_x * numThreads_y);

    // Load total number of nonce iterations (tasks)
    uint32_t tasks = d_nums[4];
    uint32_t arenasize    = d_nums[11];
    uint32_t DeadlineMs = d_nums[3];
	if ((uint32_t)(tid) >= tasks) {
		// stop unwanted threads
		return;
	}

    // Each thread processes a subset of nonce iterations using a grid–stride loop.
    for (uint32_t iterator = 0; iterator < 1 + DeadlineMs; iterator ++) {
	if (mustExit()) {
		goto exit;
	}


        // Reconstruct globals from d_nums.
	alphabet0 = alpha0;
	alphabet1 = alpha1;
        uint32_t maxx       = d_nums[0];
        uint32_t len0       = d_nums[1];
        uint32_t len1       = d_nums[2];
        //uint32_t DeadlineMs = d_nums[3];
        uint32_t center     = d_nums[6];
        // d_nums[7] is used as out_offset (in units of uint32_t)
        // d_nums[8] is minsub.
	//uint32_t phase     = d_nums[7];
        //uint32_t minsub    = d_nums[8];
        uint32_t res        = d_nums[9];
        uint32_t minadd     = d_nums[10];
        uint32_t subtractor = d_nums[15];
	//uint32_t u = 10;
	//uint32_t retry = 10;
	
	// if program offset exceeded the results, our solution wouldn't be possible
	// to be added, so we exit here
	if (atomicAdd(&d_nums[7], 0) >= res) {
		goto exit;
	}

/*
        // --- Deadline / Unstucker check ---
        if (DeadlineMs > 0 && (iterator % DeadlineMs == (DeadlineMs - 1))) {
            //minsub = nonce;
            iterator = 0;
            // If phase is 0 then we are in the initial phase.
            if (phase == 0) {
		//printf("[%d %d] %d\n", (uint32_t)tid, tasks, maxx);
		// unstucker
		maxx *= u;
		maxx /= uint32_t(retry + 1);
		if (maxx == 0) {
			goto exit;
		}
		atomicExch(&d_nums[0], maxx);
            } else {
		//printf("[%d %d] %d\n", (uint32_t)tid, tasks, maxx);
                // Otherwise, increment maxx.
		atomicAdd(&d_nums[0], 1);
		maxx++;
		if (maxx > maxmax) {
			u--;
			if (u == 0) {
				goto exit;
			}
			atomicExch(&d_nums[0], 0);
			maxx = 0;
		}
                // (You might want to compare against an upper bound and adjust here.)
            }
            // Atomically update d_nums[0] and d_nums[8] with the new maxx and minsub.
            //atomicExch(&d_nums[8], minsub);
            // Proceed to the next master loop iteration.
            continue;
        }
*/
        // --- Termination condition ---
        // If the alphabets have converged to a single element (alphabet0 = [0] and alphabet1 = [1]), exit.
        if ((len0 == 1) && (len1 == 1)) {
            if (alpha0[0] == 0 && alpha1[0] == 1) {
                goto exit;
            }
        }

        // --- Compute candidate salt ("centers") ---
        uint32_t centers = center ^ (minadd + atomicAdd((uint32_t*)(void*)&hashCounter, 1));
	
        // --- Set up per-thread scratch buffer ---
        // Each thread's scratch region is a contiguous block of bytes in d_set.
	uint8_t* buf = &d_set[(uint32_t)(tid) * scratch];
        // We use a simple scheme: our "buf" is divided into "subwords" of 1 byte each.
        // For our purposes, subwords is 4, meaning we treat each hash output v as selecting a byte and 2 bits within it.
        const uint32_t subwords = 4;


        // Zero-out the scratch buffer.
        for (uint32_t i = 0; i < scratch; i++) {
            buf[i] = 0;
        }

        // Local counters for new alphabet sizes.
        uint32_t size0 = 0, size1 = 0;
        const uint8_t twobitmask = 3;

        // Define inline lambda to update the scratch buffer for a given hash value.
        auto isvBad = [&] __device__ (uint32_t v, uint8_t which) -> bool {
            uint32_t w0 = v / subwords;
            uint32_t w1 = (v % subwords) << 1;
            uint8_t current = (buf[w0] >> w1) & twobitmask;
            if (current == 0) {
                if (which == 0)
                    size0++;
                else
                    size1++;
            }
            buf[w0] |= ((1 + which) << w1);
            // If adjacent bits are both nonzero, consider this a collision.
            return ((buf[w0] & (buf[w0] >> 1)) != 0);
        };

        // Define inline lambda to "mark" a hash value.
        auto mark = [&] __device__ (uint32_t v) -> bool {
            uint32_t w0 = v / subwords;
            uint32_t w1 = (v % subwords) << 1;
            if (((buf[w0] >> w1) & twobitmask) == twobitmask)
                return false;
            buf[w0] |= (twobitmask << w1);
            return true;
        };

	if (mustExit()) {
		goto exit;
	}

        bool skip = false;
        // Process the first min(len0,len1) entries from both alphabets.
        uint32_t minl = (len0 < len1) ? len0 : len1;
        for (uint32_t i = 0; i < minl; i++) {
            for (uint8_t j = 0; j < 2; j++) {
                uint32_t val = (j == 0) ? alpha0[i] : alpha1[i];
                uint32_t v = hash_d(val, centers, maxx);
                if (isvBad(v, j)) {
                    skip = true;
                    break;
                }
            }
            if (skip)
                break;
        }
        if (skip)
            continue;

	if (mustExit()) {
		goto exit;
	}

        // Process the remaining entries for each alphabet.
        for (uint8_t j = 0; j < 2; j++) {
            uint32_t currentLen = (j == 0) ? len0 : len1;
            for (uint32_t i = minl; i < currentLen; i++) {
                uint32_t val = (j == 0) ? alpha0[i] : alpha1[i];
                uint32_t v = hash_d(val, centers, maxx);
                if (isvBad(v, j)) {
                    skip = true;
                    break;
                }
            }
            if (skip)
                break;
        }
        if (skip)
            continue;

        // Early exit: if the total number of distinct markings is not 2
        // and equals the total number of inputs, then skip this candidate.
        if (((size0 + size1) != 2) && ((len0 + len1) == (size0 + size1)))
            continue;
	
	//printf("[%d] %d %d\n", (uint32_t)tid, centers, maxx);


	if (mustExit()) {
		goto exit;
	}

        // --- Candidate accepted: build new alphabets ---
        uint32_t win_centers = centers;
        // Temporary arrays for new alphabets.
	uint32_t add0 = allocate(size0);
        uint32_t* new_alpha0 = &arena[add0];
	if ((new_alpha0 == 0) || (size0+add0 >= arenasize)) {
		new_alpha0 = 0;
	}
	uint32_t add1 = allocate(size1);
        uint32_t* new_alpha1 = &arena[add1];
	if ((new_alpha1 == 0) || (size1+add1 >= arenasize)) {
		///free(new_alpha0);
		new_alpha1 = 0;
	}
	if ((new_alpha0 != 0) && (new_alpha1 != 0)) {
		uint32_t count0 = 0, count1 = 0;
		for (uint8_t j = 0; j < 2; j++) {
		    uint32_t currentLen = (j == 0) ? len0 : len1;
		    for (uint32_t i = 0; i < currentLen; i++) {
		        uint32_t val = (j == 0) ? alpha0[i] : alpha1[i];
		        uint32_t v = hash_d(val, centers, maxx);
		        if (mark(v)) {
		            if (j == 0)
		                new_alpha0[count0++] = v;
		            else
		                new_alpha1[count1++] = v;
		        }
		    }
		    if (skip)
		        break;
		}
		if (skip)
		    continue;
		size0 = count0;
		size1 = count1;
	}
	//

	// THE MUTEX
	if (lockMutex()) {
		///free(new_alpha0);
		///free(new_alpha1);
		continue;
	}
	if (mustExit()) {
		///free(new_alpha0);
		///free(new_alpha1);
		goto mutex_release_exit;
	}

        // --- Write a "program" entry (a pair: win_centers and maxx) ---
        // We use atomicAdd on d_nums[7] to reserve two uint32_t slots.
        uint32_t progOffset = atomicAdd(&d_nums[7], 2);
	//printf("[%d] %d %d %d %d\n", (uint32_t)tid, centers, maxx, progOffset, res);
	if (progOffset >= res) {
		///free(new_alpha0);
		///free(new_alpha1);
        	goto mutex_release_exit;
	}
	if (progOffset != 0) {
		if (atomicAdd(& out[progOffset - 1], 0) <= maxx) {
			///free(new_alpha0);
			///free(new_alpha1);
        		goto mutex_release_exit;
		}
	}
        atomicAdd(&out[progOffset], win_centers);
        atomicAdd(& out[progOffset + 1], maxx);
	if (progOffset+2 >= res) {
		///free(new_alpha0);
		///free(new_alpha1);
        	goto mutex_release_exit;
	}

	// we can't continue forcing since the arena was exhausted
	if ((new_alpha0 == 0) || (new_alpha1 == 0)) {
        	goto mutex_release_exit;
	}

        // --- Update global parameters based on new alphabets ---
        uint32_t new_maxl = (size0 > size1) ? size0 : size1;
        // Use a simple subtractor (could be replaced by a parameter from the host)
        uint32_t sub = subtractor;
        if (sub > new_maxl)
            sub = new_maxl - 1;
        // Compute newmaxx: maxx * ((new_maxl - sub)^2) / (new_maxl^2)
        uint32_t newmaxx = (uint32_t)(((uint64_t)maxx * (new_maxl - sub) * (new_maxl - sub)) /
                                      ((uint64_t)new_maxl * new_maxl));

        if (newmaxx <= 1) {
		//printf("[%d] %d <= 1\n", (uint32_t)tid, newmaxx);
		///free(new_alpha0);
		///free(new_alpha1);
            goto mutex_release_exit;
	}

        if (newmaxx >= maxx) {
		//printf("[%d] %d >= %d\n", (uint32_t)tid, newmaxx, maxx);
            //minsub = 0;
            center = win_centers;
		///free(new_alpha0);
		///free(new_alpha1);
            goto mutex_release_exit;
        } else {
            maxx = newmaxx;
            //minsub = nonce;
            center = win_centers;
        }
        if (maxx <= new_maxl)
            maxx = new_maxl;

	
        // Update the global parameters in d_nums.
        atomicExch(&d_nums[0], maxx);
        atomicExch(&d_nums[1], size0);
        atomicExch(&d_nums[2], size1);
        atomicExch(&d_nums[6], center);
        //atomicExch(&d_nums[8], minsub);

        // Overwrite the global alphabets with the new alphabets.
        // (This code assumes that count0 and count1 do not exceed the allocated sizes.)
	{
		if (alphabet0 != alpha0 && alphabet1 != alpha1) {
			///uint32_t* ptr0 = alpha0;
			///uint32_t* ptr1 = alpha1;
			alpha0 = new_alpha0;
			alpha1 = new_alpha1;
			///free(free0);
			///free(free1);
			///free0 = ptr0;
			///free1 = ptr1;
			
		} else {
			alpha0 = new_alpha0;
			alpha1 = new_alpha1;
			
		}
	};
	unlockMutex();


    } // end grid-stride loop over nonce
mutex_release_exit:
	unlockMutex();
        {
		if (alphabet0 != alpha0 && alphabet1 != alpha1) {
			///uint32_t* ptr0 = alpha0;
			///uint32_t* ptr1 = alpha1;
			alpha0 = alphabet0;
			alpha1 = alphabet1;
			///free(free0);
			///free(free1);
			///free0 = ptr0;
			///free1 = ptr1;

			
		}
	};
exit:
	//printf("[%d] Timeouted\n", (uint32_t)tid);
	// once some thread expires the timeout, all threads must quit
	//cudaFree(data0); cudaFree(data1);
	///free(free0);
	///free(free1);
	///free0 = 0;
	///free1 = 0;
	
	atomicAdd((uint32_t *)(void *)&d_nums[13], myFlag+1);
	
	return;

}

