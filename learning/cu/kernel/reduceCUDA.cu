#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>



__device__ uint32_t hash(uint32_t n, uint32_t s, uint32_t max) {
	// mixing stage, mix input with salt using subtraction
	// (could also be addition)
	uint32_t m = n - s;

	// hashing stage, use xor shift with prime coefficients
	m ^= m << 2;
	m ^= m << 3;
	m ^= m >> 5;
	m ^= m >> 7;
	m ^= m << 11;
	m ^= m << 13;
	m ^= m >> 17;
	m ^= m << 19;

	// mixing stage 2, mix input with salt using addition
	m += s;

	// modular stage
	// to force output in range 0 to max-1 we could do regular modulo
	// however, the faster multiply shift trick by Daniel Lemire is used instead
	// https://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
	return (uint32_t)(((uint64_t)m * (uint64_t)max) >> 32);
}


__device__ uint32_t real_modulo_recip(uint32_t y) {
	return uint32_t((uint64_t(1) << 32) / uint64_t(y));
}

__device__ uint32_t real_modulo(uint32_t x, uint32_t recip, uint32_t y) {
	return uint32_t((uint64_t(uint32_t((x + 1) * recip)) * uint64_t(y)) >> 32);
}

__device__ static int exitFlag = 0;


extern "C" __global__ void reduce(uint8_t *d_set, uint32_t *d_nums, uint32_t *alphabet, uint32_t* out) {
	uint32_t max = d_nums[0];
	uint32_t maxl = d_nums[1];
	uint32_t timeMs = d_nums[2];
	uint32_t tasks = d_nums[3];
	uint32_t iteration = d_nums[4];
	uint32_t center = d_nums[5];

	int myFlag = iteration;
	uint64_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
	uint64_t tid_z = blockIdx.z * blockDim.z + threadIdx.z;

	uint64_t numThreads_x = gridDim.x * blockDim.x;
	uint64_t numThreads_y = gridDim.y * blockDim.y;

	uint64_t tid = tid_x + tid_y * numThreads_x + tid_z * (numThreads_x * numThreads_y);

	uint32_t s = tid;

	if (s >= tasks) {
		// stop unwanted threads
		return;
	}

	uint32_t maxl_recip = real_modulo_recip(maxl);
	clock_t start = clock();

	for (; (((clock() - start) / (float)CLOCKS_PER_SEC) < (((float)timeMs)*(float)0.001f)); s += tasks) {
		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		//__syncthreads();
		if (maxl > 4) {
			uint8_t* set = &d_set[tid * (((max + 3) / 4) + 4)];
			uint32_t i = 0;
			uint32_t v = alphabet[i];
			uint32_t size = 0;
			for (uint32_t j = 0; j < 2 * maxl; j++) {
				i = hash(v, center^s, max);
				v = alphabet[(j & 1) * maxl + uint32_t(((uint64_t)((i + 1) * maxl_recip) * uint64_t(maxl)) >> 32)];

				uint32_t imodmax = i;
				if ((set[imodmax >> 2] >> ((imodmax & 3) << 1)) & 3 != 0) {
					if ((set[imodmax >> 2] >> ((imodmax & 3) << 1)) & 3 == ((j ^ 1) & 1) + 1) {
						goto next_iteration;
					}
				} else {
					size++;
				}
				set[imodmax >> 2] |= (j & 1) + 1 << ((imodmax & 3) << 1);
			}
			if (atomicAdd(&exitFlag, 0) > myFlag) {
				return;
			}
			if (size == 2*maxl) {
				goto next_iteration;
			}
			if (atomicAdd(&exitFlag, 0) > myFlag) {
				return;
			}
			//__syncthreads();
		}
		for (uint32_t i = 0; i < maxl; i++) {
			uint32_t v = hash(alphabet[i], center^s, max);
			for (uint32_t j = 0; j < maxl; j++) {
				uint32_t w = hash(alphabet[maxl+j], center^s, max);
					if (v == w) {
						goto next_iteration;
				}
			}
		}
		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		//__syncthreads();
		// Atomic operations to update output
		out[0] = center^s;
		out[1] = max;
		atomicExch(&exitFlag, myFlag+1);
		//__syncthreads();

		return;

		next_iteration:
		{
			uint8_t* set = &d_set[tid * (((max + 3) / 4) + 4)];
			for (uint32_t i = 0; i < ((max + 3) / 4); i++) {
				set[i] = 0;
			}
		}
	}
}
