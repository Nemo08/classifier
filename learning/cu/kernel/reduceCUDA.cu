#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>



__device__ uint32_t hash(uint32_t n, uint32_t s, uint32_t max) {
	// mixing stage, mix input with salt using subtraction
	// (could also be addition)
	uint32_t m = n - s;

	// hashing stage, use xor shift with prime coefficients
	m ^= m << 2;
	m ^= m << 3;
	m ^= m >> 5;
	m ^= m >> 7;
	m ^= m << 11;
	m ^= m << 13;
	m ^= m >> 17;
	m ^= m << 19;

	// mixing stage 2, mix input with salt using addition
	m += s;

	// modular stage
	// to force output in range 0 to max-1 we could do regular modulo
	// however, the faster multiply shift trick by Daniel Lemire is used instead
	// https://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
	return (uint32_t)(((uint64_t)m * (uint64_t)max) >> 32);
}


__device__ uint32_t real_modulo_recip(uint32_t y) {
	return uint32_t((uint64_t(1) << 32) / uint64_t(y));
}

__device__ uint32_t real_modulo(uint32_t x, uint32_t recip, uint32_t y) {
	return uint32_t((uint64_t(uint32_t((x + 1) * recip)) * uint64_t(y)) >> 32);
}

__device__ int exitFlag = 0;


extern "C" __global__ void reduce(uint8_t *d_set, uint32_t max, uint32_t maxl, uint32_t *alphabet, uint32_t* out) {
	int myFlag = atomicAdd(&exitFlag, 0);
	__syncthreads();
	uint32_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t tid_z = blockIdx.z * blockDim.z + threadIdx.z;

	uint32_t numThreads_x = gridDim.x * blockDim.x;
	uint32_t numThreads_y = gridDim.y * blockDim.y;
	uint32_t numThreads_z = gridDim.z * blockDim.z;

	uint32_t tid = tid_x + tid_y * numThreads_x + tid_z * (numThreads_x * numThreads_y);

	uint32_t totalThreads = numThreads_x * numThreads_y * numThreads_z;
	uint32_t s = tid;
	uint32_t maxl_recip = real_modulo_recip(maxl);
	clock_t start = clock();

	for (; ((clock() - start) / (float)CLOCKS_PER_SEC < 0.1f); s += totalThreads) {
		if (atomicAdd(&exitFlag, 0) != myFlag) {
			return;
		}
		//__syncthreads();
		if (maxl > 4) {
			uint8_t* set = &d_set[tid * ((max + 3) / 4)];
			for (uint32_t i = 0; i < ((max + 3) / 4); i++) {
				set[i] = 0;
			}
			uint32_t i = 0;
			uint32_t v = alphabet[i];
			for (uint32_t j = 0; j < 2 * maxl; j++) {
				i = hash(v, s, max);
				v = alphabet[(j & 1) * maxl + uint32_t(((uint64_t)((i + 1) * maxl_recip) * uint64_t(maxl)) >> 32)];

				uint32_t imodmax = i;
				if ((set[imodmax >> 2] >> ((imodmax & 3) << 1)) & 3 != 0) {
					if ((set[imodmax >> 2] >> ((imodmax & 3) << 1)) & 3 == ((j ^ 1) & 1) + 1) {
						goto next_iteration;
					}
				}
				set[imodmax >> 2] |= (j & 1) + 1 << ((imodmax & 3) << 1);
			}
			if (atomicAdd(&exitFlag, 0) != myFlag) {
				return;
			}
			//__syncthreads();
		}
		for (uint32_t i = 0; i < maxl; i++) {
			uint32_t v = hash(alphabet[i], s, max);
			for (uint32_t j = 0; j < maxl; j++) {
				uint32_t w = hash(alphabet[maxl+j], s, max);
					if (v == w) {
						goto next_iteration;
				}
			}
		}
		if (atomicAdd(&exitFlag, 0) != myFlag) {
			return;
		}
		//__syncthreads();
		// Atomic operations to update output
		out[0] = s;
		out[1] = max;
		atomicAdd(&exitFlag, 1);
		//__syncthreads();

		return;

		next_iteration:;
	}
}

extern "C" void reduceCUDA(uint32_t blk, uint32_t grid, uint32_t max, uint32_t maxl,
				uint32_t *alphabet, uint32_t* result0, uint32_t* result1)
{
	uint32_t result[2] = {0,0};

	uint32_t *d_input;
	uint32_t *d_result;
	uint8_t *d_set;

	int blockSize = (int)blk;
	int gridSize = (int)grid;
	hipMalloc((void**)&d_input, sizeof(uint32_t) * maxl * 2);
	hipMalloc((void**)&d_result, sizeof(uint32_t) * 2);
	hipMalloc((void**)&d_set, sizeof(uint8_t) * blockSize * gridSize);
	hipMemcpy(d_input, alphabet, sizeof(uint32_t) * maxl * 2, hipMemcpyHostToDevice);
	reduce<<<gridSize, blockSize>>>(d_set, max, maxl, d_input, d_result);
	hipMemcpy(&result, d_result, sizeof(uint32_t) * 2, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipFree(d_result);
	hipFree(d_set);

	*result0 = result[0];
	*result1 = result[1];

	return;
}
