#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>



__device__ uint32_t hash(uint32_t n, uint32_t s, uint32_t max) {
	// mixing stage, mix input with salt using subtraction
	// (could also be addition)
	uint32_t m = n - s;

	// hashing stage, use xor shift with prime coefficients
	m ^= m << 2;
	m ^= m << 3;
	m ^= m >> 5;
	m ^= m >> 7;
	m ^= m << 11;
	m ^= m << 13;
	m ^= m >> 17;
	m ^= m << 19;

	// mixing stage 2, mix input with salt using addition
	m += s;

	// modular stage
	// to force output in range 0 to max-1 we could do regular modulo
	// however, the faster multiply shift trick by Daniel Lemire is used instead
	// https://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
	return (uint32_t)(((uint64_t)m * (uint64_t)max) >> 32);
}


__device__ uint32_t real_modulo_recip(uint32_t y) {
	return uint32_t((uint64_t(1) << 32) / uint64_t(y));
}

__device__ uint32_t real_modulo(uint32_t x, uint32_t recip, uint32_t y) {
	return uint32_t((uint64_t(uint32_t((x + 1) * recip)) * uint64_t(y)) >> 32);
}

__device__ static int exitFlag = 0;


extern "C" __global__ void reduce(uint8_t *d_set, uint32_t *d_nums, uint32_t *alphabet0, uint32_t *alphabet1, uint32_t* out) {
	uint32_t max = d_nums[0];
	uint32_t l0 = d_nums[1];
	uint32_t l1 = d_nums[2];
	uint32_t timeMs = d_nums[3];
	uint32_t tasks = d_nums[4];
	uint32_t iteration = d_nums[5];
	uint32_t center = d_nums[6];
	uint32_t *alphabet[2] = {alphabet0, alphabet1};
	uint32_t minl = l0;
	if (l1 < l0) {
		minl = l1;
	}

	int myFlag = iteration;
	uint64_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
	uint64_t tid_z = blockIdx.z * blockDim.z + threadIdx.z;

	uint64_t numThreads_x = gridDim.x * blockDim.x;
	uint64_t numThreads_y = gridDim.y * blockDim.y;

	uint64_t tid = tid_x + tid_y * numThreads_x + tid_z * (numThreads_x * numThreads_y);

	uint32_t s = tid;

	if (s >= tasks) {
		// stop unwanted threads
		return;
	}

	clock_t start = clock();

	for (; (((clock() - start) / (float)CLOCKS_PER_SEC) < (((float)timeMs)*(float)0.001f)); s += tasks) {
		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		//__syncthreads();
		uint8_t* set = &d_set[tid * (((max + 3) / 4) + 4)];
		uint32_t size = 0;
		for (uint32_t j = 0; j < minl; j++) {
			//if (atomicAdd(&exitFlag, 0) > myFlag) {
			//	return;
			//}
			for (uint8_t jj = 0; jj < 2; jj++) {
				uint32_t i = alphabet[jj][j];
				uint32_t v = hash(i, center^s, max);
				const uint8_t subwords = 4;
				const uint8_t twobitmask = 3;
				uint32_t w0 = v / subwords;
				uint32_t w1 = (v % subwords) << 1;
				uint8_t loaded = (set[w0] >> w1) & twobitmask;
				if (loaded == (2 - jj)) {
					goto next_iteration;
				}
				if (loaded == 0) {
					size++;
				}
				set[w0] |= ((1 + jj) << w1);
			}
		}

		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		for (uint32_t j = minl; j < l0; j++) {
			//if (atomicAdd(&exitFlag, 0) > myFlag) {
			//	return;
			//}
			uint8_t jj = 0;
			uint32_t i = alphabet[jj][j];
			uint32_t v = hash(i, center^s, max);
			const uint8_t subwords = 4;
			const uint8_t twobitmask = 3;
			uint32_t w0 = v / subwords;
			uint32_t w1 = (v % subwords) << 1;
			uint8_t loaded = (set[w0] >> w1) & twobitmask;
			if (loaded == (2 - jj)) {
				goto next_iteration;
			}
			if (loaded == 0) {
				size++;
			}
			set[w0] |= (1 + jj) << w1;
		}
		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		for (uint32_t j = minl; j < l1; j++) {
			//if (atomicAdd(&exitFlag, 0) > myFlag) {
			///	return;
			//}
			uint8_t jj = 1;
			uint32_t i = alphabet[jj][j];
			uint32_t v = hash(i, center^s, max);
			const uint8_t subwords = 4;
			const uint8_t twobitmask = 3;
			uint32_t w0 = v / subwords;
			uint32_t w1 = (v % subwords) << 1;
			uint8_t loaded = (set[w0] >> w1) & twobitmask;
			if (loaded == (2 - jj)) {
				goto next_iteration;
			}
			if (loaded == 0) {
				size++;
			}
			set[w0] |= (1 + jj) << w1;
		}

		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		if (size == l0 + l1) {
			goto next_iteration;
		}
		if (atomicAdd(&exitFlag, 0) > myFlag) {
			return;
		}
		//__syncthreads();
		//__syncthreads();
		// Atomic operations to update output
		out[0] = center^s;
		out[1] = max;
		atomicExch(&exitFlag, myFlag+1);
		//__syncthreads();
		return;

		next_iteration:
		{
			uint8_t* set = &d_set[tid * (((max + 3) / 4) + 4)];
			for (uint32_t i = 0; i < ((max + 3) / 4) + 4; i++) {
				set[i] = 0;
			}
		}
	}
}
